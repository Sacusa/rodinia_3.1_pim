#include <cassert>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <thread>
#include <vector>
#include "../../common.h"

/**
 * std::barrier and std::semaphore support was added in C++20 and requires
 * CUDA 12+. We provide comparable implementations for older versions of CUDA,
 * such as used the one we use. This requires C++11 support.
 */

#if __cplusplus >= 202002L

#include <barrier>
#include <semaphore>
using Barrier = std::barrier;
using Semaphore = std::counting_semaphore;

#else

#include <condition_variable>
#include <mutex>

class Barrier {
    public:
        Barrier (unsigned expected_ = 0) : expected(expected_) {}

        inline void arrive_and_wait() {
            std::unique_lock<std::mutex> lock(mtx);
            --expected;
            if (expected == 0) {
                lock.unlock();
                cv.notify_all();
            } else {
                cv.wait(lock, [&] { return expected == 0; });
            }
        }

    private:
        std::mutex mtx;
        std::condition_variable cv;
        unsigned expected;
};

class Semaphore {
    public:
        Semaphore (unsigned count_ = 0) : count(count_) {}

        inline void acquire() {
            std::unique_lock<std::mutex> lock(mtx);
            if (count == 0) {
                cv.wait(lock, [&] { return count > 0; });
            }
            --count;
        }

        inline bool try_acquire() {
            std::unique_lock<std::mutex> lock(mtx);
            if (count == 0) {
                return false;
            } else {
                --count;
                return true;
            }
        }

        inline void release() {
            std::unique_lock<std::mutex> lock(mtx);
            ++count;
            lock.unlock();
            cv.notify_all();
        }

private:
        std::mutex mtx;
        std::condition_variable cv;
        unsigned count;
};

#endif

#define MIN_ITERS 1
#define PIM_ROWS  256
#define PIM_SMS   8  // maximum number of SMs apportioned for PIM kernels
#define LLM_NUM_MEM_KERNELS 3

enum ExecMode {
    MEM_ONLY = 0,
    PIM_ONLY,
    MEM_AND_MEM,
    PIM_AND_MEM,
    LLM,
    LLM_MEM_ONLY,
    LLM_PIM_ONLY,
    NUM_EXEC_MODES
};

std::vector<std::string> exec_mode_str {
    "MEM only",
    "PIM only",
    "MEM and MEM",
    "PIM and MEM",
    "LLM",
    "LLM (MEM only)",
    "LLM (PIM only)"
};

// Global state variables
int (*mem_app[2]) (int, char**);
hipStream_t mem1_stream;

pim_state_t *pim_state;
hipStream_t pim_stream;

// Rodinia benchmark declarations
extern "C" int main_btree(int argc, char** argv);
int main_backprop(int argc, char** argv);
int main_bfs(int argc, char** argv);
int main_euler3d(int argc, char** argv);  // CFD
int main_dwt2d(int argc, char** argv);
int main_gaussian(int argc, char** argv);
int main_heartwall(int argc, char** argv);
int main_hotspot(int argc, char** argv);
int main_hotspot3D(int argc, char** argv);
int main_huffman(int argc, char** argv);
int main_kmeans(int argc, char** argv, hipStream_t stream);
int main_lavaMD(int argc, char** argv);
extern "C" int main_leukocyte(int argc, char** argv);
int main_lud(int argc, char** argv);
int main_mummergpu(int argc, char** argv);
int main_nn(int argc, char** argv, bool is_first);
int main_nw(int argc, char** argv);
int main_pathfinder(int argc, char** argv);
int main_srad_v1(int argc, char** argv);
int main_srad_v2(int argc, char** argv);
int main_streamcluster(int argc, char** argv);
void main_gemm(hipStream_t stream, size_t M, size_t K, size_t N);

// Helper functions
char **parse_arguments(char *args, int &argc);
void setup_mem(char*, int);
void setup_pim(char *kernel);
void run_mem(int, char*, int, char**, bool, bool, bool, Barrier*, Semaphore*,
        Semaphore*, Semaphore*, Semaphore*);
void run_gemm(hipStream_t, bool, Barrier*, Semaphore*, Semaphore*,
        Semaphore*);
void run_pim(bool, Barrier*, Semaphore*, Semaphore*, Semaphore*);
void run_pim_llm(pim_state_t*, pim_state_t*, pim_state_t*, Barrier*,
        Semaphore*, Semaphore*, Semaphore*);
void exec_mem_only(int, char**);
void exec_pim_only(int, char**);
void exec_mem_and_mem(int, char**, int, char**);
void exec_pim_and_mem(int, char**, int, char**);
void exec_llm(bool, bool);

void print_usage(char *argv0)
{
    std::cout << "Usage: " << argv0 << \
        " <mode> \"[pim/mem1 [args]]\" \"[mem/mem2 [args]]\"" << std::endl;
    std::cout << std::endl;

    std::cout << "Modes:" << std::endl;

    for (int exec_mode_iter = 0; exec_mode_iter != NUM_EXEC_MODES;
            exec_mode_iter++) {
        std::cout << exec_mode_iter << "\t" << \
            exec_mode_str[exec_mode_iter] << std::endl;
    }
}

int main(int argc, char **argv)
{
    enum args_positions {
        BIN = 0,
        MODE,
        APP1,
        APP2,
        NUM_ARGS
    };

    int error_code = EXIT_SUCCESS;

    if (argc <= MODE) {
        print_usage(argv[BIN]);
        error_code = EXIT_FAILURE;
    }

    else {
        int exec_mode_int = std::atoi(argv[MODE]);

        if (exec_mode_int >= NUM_EXEC_MODES) {
            print_usage(argv[BIN]);
            error_code = EXIT_FAILURE;
        }

        ExecMode exec_mode = static_cast<ExecMode>(exec_mode_int);

        if (exec_mode == MEM_ONLY) {
            if (argc != (APP1 + 1)) {
                print_usage(argv[BIN]);
                error_code = EXIT_FAILURE;
            } else {
                int mem_argc = 0;
                char **mem_argv = parse_arguments(argv[APP1], mem_argc);

                exec_mem_only(mem_argc, mem_argv);
            }
        }

        else if (exec_mode == PIM_ONLY) {
            if (argc != (APP1 + 1)) {
                print_usage(argv[BIN]);
                error_code = EXIT_FAILURE;
            } else {
                int pim_argc = 0;
                char **pim_argv = parse_arguments(argv[APP1], pim_argc);

                exec_pim_only(pim_argc, pim_argv);
            }
        }

        else if (exec_mode == MEM_AND_MEM) {
            if (argc != (APP2 + 1)) {
                print_usage(argv[BIN]);
                error_code = EXIT_FAILURE;
            } else {
                int mem1_argc = 0;
                char **mem1_argv = parse_arguments(argv[APP1], mem1_argc);

                int mem2_argc = 0;
                char **mem2_argv = parse_arguments(argv[APP2], mem2_argc);

                exec_mem_and_mem(mem1_argc, mem1_argv, mem2_argc, mem2_argv);
            }
        }

        else if (exec_mode == PIM_AND_MEM) {
            if (argc != (APP2 + 1)) {
                print_usage(argv[BIN]);
                error_code = EXIT_FAILURE;
            } else {
                int pim_argc = 0;
                char **pim_argv = parse_arguments(argv[APP1], pim_argc);

                int mem_argc = 0;
                char **mem_argv = parse_arguments(argv[APP2], mem_argc);

                exec_pim_and_mem(pim_argc, pim_argv, mem_argc, mem_argv);
            }
        }

        else if (exec_mode == LLM) {
            if (argc != (MODE + 1)) {
                print_usage(argv[BIN]);
                error_code = EXIT_FAILURE;
            } else {
                exec_llm(true, true);
            }
        }

        else if (exec_mode == LLM_MEM_ONLY) {
            if (argc != (MODE + 1)) {
                print_usage(argv[BIN]);
                error_code = EXIT_FAILURE;
            } else {
                exec_llm(true, false);
            }
        }

        else if (exec_mode == LLM_PIM_ONLY) {
            if (argc != (MODE + 1)) {
                print_usage(argv[BIN]);
                error_code = EXIT_FAILURE;
            } else {
                exec_llm(false, true);
            }
        }

        else {
            // Should not reach here
            std::cout << "Unknown mode " << exec_mode << std::endl;
            error_code = EXIT_FAILURE;
        }
    }

    return error_code;
}

/**
 * This function converts a string with an application name and associated
 * arguments into a C-style argv array. It also returns argc using the second
 * argument.
 */
char **parse_arguments(char *args, int &argc)
{
    std::string args_str(args);
    std::istringstream args_tokens(args_str);
    std::vector<char*> args_vector;

    argc = 0;

    // Stream the argument tokens into a char* vector
    std::string token;
    while (getline(args_tokens, token, ' ')) {
        size_t token_size = token.size();

        char *arg = (char*) malloc(sizeof(char) * (token_size + 1));
        token.copy(arg, token_size);
        arg[token_size] = '\0';

        args_vector.push_back(arg);
        argc++;
    }

    assert(argc > 0);  // at least the application name should be there

    char **argv = (char**) malloc(sizeof(char*) * (argc + 1));
    std::copy(args_vector.begin(), args_vector.end(), argv);
    argv[argc] = NULL;

    return argv;
}

void setup_mem(char *kernel, int index)
{
    if (!strcmp(kernel, "b+tree")) {
        mem_app[index] = main_btree;
    } else if (!strcmp(kernel, "backprop")) {
        mem_app[index] = main_backprop;
    } else if (!strcmp(kernel, "bfs")) {
        mem_app[index] = main_bfs;
    } else if (!strcmp(kernel, "cfd")) {
        mem_app[index] = main_euler3d;
    } else if (!strcmp(kernel, "dwt2d")) {
        mem_app[index] = main_dwt2d;
    } else if (!strcmp(kernel, "gaussian")) {
        mem_app[index] = main_gaussian;
    } else if (!strcmp(kernel, "heartwall")) {
        mem_app[index] = main_heartwall;
    } else if (!strcmp(kernel, "hotspot")) {
        mem_app[index] = main_hotspot;
    } else if (!strcmp(kernel, "hotspot3D")) {
        mem_app[index] = main_hotspot3D;
    } else if (!strcmp(kernel, "huffman")) {
        mem_app[index] = main_huffman;
    } else if (!strcmp(kernel, "kmeans")) {
        // do nothing; kmeans needs a stream argument
    } else if (!strcmp(kernel, "lavaMD")) {
        mem_app[index] = main_lavaMD;
    } else if (!strcmp(kernel, "leukocyte")) {
        mem_app[index] = main_leukocyte;
    } else if (!strcmp(kernel, "lud")) {
        mem_app[index] = main_lud;
    } else if (!strcmp(kernel, "mummergpu")) {
        mem_app[index] = main_mummergpu;
    } else if (!strcmp(kernel, "nn")) {
        // do nothing; nn needs special handling
    } else if (!strcmp(kernel, "nw")) {
        mem_app[index] = main_nw;
    } else if (!strcmp(kernel, "pathfinder")) {
        mem_app[index] = main_pathfinder;
    } else if (!strcmp(kernel, "srad_v1")) {
        mem_app[index] = main_srad_v1;
    } else if (!strcmp(kernel, "srad_v2")) {
        mem_app[index] = main_srad_v2;
    } else if (!strcmp(kernel, "streamcluster")) {
        mem_app[index] = main_streamcluster;
    } else {
        std::cout << "Invalid MEM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void setup_pim(char *kernel)
{
    if (!strcmp(kernel, "stream_add")) {
        pim_state = init_pim(STREAM_ADD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_copy")) {
        pim_state = init_pim(STREAM_COPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_daxpy")) {
        pim_state = init_pim(STREAM_DAXPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_scale")) {
        pim_state = init_pim(STREAM_SCALE, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_triad")) {
        pim_state = init_pim(STREAM_TRIAD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_fwd")) {
        pim_state = init_pim(BN_FWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_bwd")) {
        pim_state = init_pim(BN_BWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "kmeans")) {
        pim_state = init_pim(KMEANS, 1048576, 1);
    } else if (!strcmp(kernel, "histogram")) {
        pim_state = init_pim(HISTOGRAM, 1048576, 1);
    } else if (!strcmp(kernel, "fully_connected")) {
        pim_state = init_pim(FULLY_CONNECTED, 1048576, 256);
    } else if (!strcmp(kernel, "grim")) {
        pim_state = init_pim(GRIM, 1048576, 32);
    } else {
        std::cout << "Invalid PIM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void run_mem(int mem_app_index, char *kernel, int argc, char **argv,
        bool is_first, bool do_wait_to_launch, bool do_signal_launch,
        Barrier *all_threads_ready_barrier, Semaphore *signal_mem_start,
        Semaphore *mem_running, Semaphore *thread_finished,
        Semaphore *signal_mem_finished)
{
    // create a copy of argv because some benchmarks destroy the arguments
    char **argv_copy = new char*[argc + 1];
    for (int i = 0; i < argc; i++) {
        argv_copy[i] = new char[strlen(argv[i]) + 1];
        strcpy(argv_copy[i], argv[i]);
    }
    argv_copy[argc] = NULL;

    // ensure all threads start at the same time
    if (is_first) { all_threads_ready_barrier->arrive_and_wait(); }

    if (do_wait_to_launch) { signal_mem_start->acquire(); }
    if (do_signal_launch)  { mem_running->release(); }

    hipStream_t stream;
    switch (mem_app_index) {
        case 0:
            stream = 0;  // default stream
            break;
        case 1:
            stream = mem1_stream;
            break;
        default:
            std::cout << "Invalid mem app index" << std::endl;
            exit(EXIT_FAILURE);
    }

    // Only kmeans uses the passed stream object for now. This can be changed
    // but it will take time.
    if (!strcmp(kernel, "kmeans")) {
        main_kmeans(argc, argv_copy, stream);
    } else if (!strcmp(kernel, "nn")) {
        main_nn(argc, argv_copy, is_first);
    } else {
        mem_app[mem_app_index](argc, argv_copy);
    }

    hipStreamSynchronize(stream);
    signal_mem_finished->release();
    thread_finished->release();

    // we don't need to destroy argv_copy because:
    // 1) if the benchmark modified the pointer, freeing it can cause a
    //    segfault
    // 2) the thread is being destroyed right now, so the memory will be freed
    //    anyway
}

void run_gemm(hipStream_t stream, bool wait_for_pim,
        Barrier *all_threads_ready_barrier, Semaphore *pim_running,
        Semaphore *thread_finished, Semaphore *signal_mem_finished)
{
    all_threads_ready_barrier->arrive_and_wait();

    if (wait_for_pim) { pim_running->acquire(); }

    for (int i = 0; i < LLM_NUM_MEM_KERNELS; i++) {
        main_gemm(stream, 128, 4096, 4096);
        hipStreamSynchronize(stream);
    }

    signal_mem_finished->release();
    thread_finished->release();
}

void run_pim(bool is_first, Barrier *all_threads_ready_barrier,
        Semaphore *pim_running, Semaphore *thread_finished,
        Semaphore *signal_pim_finished)
{
    if (is_first) { all_threads_ready_barrier->arrive_and_wait(); }

    launch_pim(pim_state, pim_stream);

    if (is_first) { pim_running->release(); }

    hipStreamSynchronize(pim_stream);
    signal_pim_finished->release();
    thread_finished->release();
}

void run_pim_llm(pim_state_t *pim_qk, pim_state_t *pim_softmax,
        pim_state_t *pim_sv, Barrier *all_threads_ready_barrier,
        Semaphore *pim_running, Semaphore *thread_finished,
        Semaphore *signal_pim_finished)
{
    all_threads_ready_barrier->arrive_and_wait();

    launch_pim(pim_qk,      pim_stream);
    launch_pim(pim_softmax, pim_stream);
    launch_pim(pim_sv,      pim_stream);

    pim_running->release();

    hipStreamSynchronize(pim_stream);
    signal_pim_finished->release();
    thread_finished->release();
}

void exec_mem_only(int argc, char **argv)
{
    char *mem_app_name = argv[0];
    setup_mem(mem_app_name, 0);

    unsigned mem_iters = 0;
    Semaphore thread_finished{0}, mem_finished{0};
    Barrier all_threads_ready_barrier(1);

    while (mem_iters < MIN_ITERS) {
        std::thread (run_mem, 0, mem_app_name, argc, argv, mem_iters == 0,
                false, false, &all_threads_ready_barrier, nullptr, nullptr,
                &thread_finished, &mem_finished).detach();

        thread_finished.acquire();

        mem_finished.acquire();
        mem_iters++;
        std::cout << "<<< MEM FINISHED >>>" << std::endl;
        hipGetErrorName(hipSuccess);
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
}

void exec_pim_only(int argc, char **argv)
{
    char *pim_app_name = argv[0];
    setup_pim(pim_app_name);

    unsigned pim_iters = 0;
    Semaphore pim_launched{0}, thread_finished{0}, pim_finished{0};
    Barrier all_threads_ready_barrier(1);

    while (pim_iters < MIN_ITERS) {
        std::thread (run_pim, false, &all_threads_ready_barrier,
                &pim_launched, &thread_finished, &pim_finished).detach();

        thread_finished.acquire();

        pim_finished.acquire();
        pim_iters++;
        std::cout << "<<< PIM FINISHED >>>" << std::endl;
        hipGetErrorName(hipSuccess);
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
}

void exec_mem_and_mem(int mem1_argc, char **mem1_argv, int mem2_argc,
        char **mem2_argv)
{
    char *mem1_app_name = mem1_argv[0];
    char *mem2_app_name = mem2_argv[0];
    setup_mem(mem1_app_name, 0);
    setup_mem(mem2_app_name, 1);

    hipStreamCreateWithPriority(&mem1_stream, PIM_SMS, -1);

    unsigned mem1_iters = 0, mem2_iters = 0;
    bool mem1_running = false, mem2_running = false;
    Semaphore mem1_launched{0}, thread_finished{0}, mem1_finished{0},
              mem2_finished{0};
    Barrier all_threads_ready_barrier(2);

    while ((mem1_iters < MIN_ITERS) || (mem2_iters < MIN_ITERS)) {
        if (!mem1_running && !mem2_running) {
            std::thread (run_mem, 0, mem1_app_name, mem1_argc, mem1_argv, true,
                    false, true, &all_threads_ready_barrier, nullptr,
                    &mem1_launched, &thread_finished, &mem1_finished).detach();
            std::thread (run_mem, 1, mem2_app_name, mem2_argc, mem2_argv, true,
                    true, false, &all_threads_ready_barrier, &mem1_launched,
                    nullptr, &thread_finished, &mem2_finished).detach();

            mem1_running = true;
            mem2_running = true;
        }

        else if (!mem1_running) {
            std::thread (run_mem, 0, mem1_app_name, mem1_argc, mem1_argv,
                    false, false, false, &all_threads_ready_barrier, nullptr,
                    &mem1_launched, &thread_finished, &mem1_finished).detach();
            mem1_running = true;
        }

        else if (!mem2_running) {
            std::thread (run_mem, 1, mem2_app_name, mem2_argc, mem2_argv,
                    false, false, false, &all_threads_ready_barrier,
                    &mem1_launched, nullptr, &thread_finished,
                    &mem2_finished).detach();
            mem2_running = true;
        }

        thread_finished.acquire();

        if (mem1_finished.try_acquire()) {
            mem1_iters++;
            mem1_running = false;
            std::cout << "<<< MEM1 FINISHED >>>" << std::endl;
            hipGetErrorName(hipSuccess);
        }

        else if (mem2_finished.try_acquire()) {
            mem2_iters++;
            mem2_running = false;
            std::cout << "<<< MEM2 FINISHED >>>" << std::endl;
            hipGetErrorName(hipSuccess);
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
}

void exec_pim_and_mem(int pim_argc, char **pim_argv, int mem_argc,
        char **mem_argv)
{
    char *pim_app_name = pim_argv[0];
    char *mem_app_name = mem_argv[0];
    setup_mem(mem_app_name, 0);
    setup_pim(pim_app_name);

    hipStreamCreateWithPriority(&pim_stream, 0, -1);

    unsigned mem_iters = 0, pim_iters = 0;
    bool mem_running = false, pim_running = false;
    Semaphore pim_launched{0}, thread_finished{0}, mem_finished{0},
              pim_finished{0};
    Barrier all_threads_ready_barrier(2);

    while ((mem_iters < MIN_ITERS) || (pim_iters < MIN_ITERS)) {
        if (!mem_running && !pim_running) {
            std::thread (run_pim, true, &all_threads_ready_barrier,
                    &pim_launched, &thread_finished, &pim_finished).detach();
            std::thread (run_mem, 0, mem_app_name, mem_argc, mem_argv, true,
                    true, false, &all_threads_ready_barrier, &pim_launched,
                    nullptr, &thread_finished, &mem_finished).detach();

            mem_running = true;
            pim_running = true;
        }

        else if (!mem_running) {
            std::thread (run_mem, 0, mem_app_name, mem_argc, mem_argv, false,
                    false, false, &all_threads_ready_barrier, &pim_launched,
                    nullptr, &thread_finished, &mem_finished).detach();
            mem_running = true;
        }

        else if (!pim_running) {
            std::thread (run_pim, false, &all_threads_ready_barrier,
                    &pim_launched, &thread_finished, &pim_finished).detach();
            pim_running = true;
        }

        thread_finished.acquire();

        if (mem_finished.try_acquire()) {
            mem_iters++;
            mem_running = false;
            std::cout << "<<< MEM FINISHED >>>" << std::endl;
            hipGetErrorName(hipSuccess);
        }

        else if (pim_finished.try_acquire()) {
            pim_iters++;
            pim_running = false;
            std::cout << "<<< PIM FINISHED >>>" << std::endl;
            hipGetErrorName(hipSuccess);
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));

    free_pim(pim_state);
}

void exec_llm(bool do_run_mem, bool do_run_pim)
{
    assert(do_run_mem || do_run_pim);

    pim_state_t *pim_qk, *pim_softmax, *pim_sv;
    hipStream_t mem_stream;

    if (do_run_pim) {
        pim_qk = init_pim(FULLY_CONNECTED, 1048576, 1024);
        pim_softmax = init_pim(SOFTMAX, 1048576, 128);
        pim_sv = init_pim(FULLY_CONNECTED_128_ELEM, 1048576, 1024);

        // Higher priority stream for PIM
        hipStreamCreateWithPriority(&pim_stream, 0, -1);
    }

    if (do_run_mem) {
        // Regular priority stream for MEM
        hipStreamCreate(&mem_stream);
    }

    Semaphore pim_launched{0}, thread_finished{0}, mem_finished{0},
              pim_finished{0};
    Barrier all_threads_ready_barrier(((do_run_pim && do_run_mem) ? 2 : 1));

    if (do_run_pim) {
        std::thread (run_pim_llm, pim_qk, pim_softmax, pim_sv,
                &all_threads_ready_barrier, &pim_launched, &thread_finished,
                &pim_finished).detach();
    }

    if (do_run_mem) {
        std::thread (run_gemm, mem_stream, do_run_pim,
                &all_threads_ready_barrier, &pim_launched, &thread_finished,
                &mem_finished).detach();
    }

    bool pim_running = do_run_pim;
    bool mem_running = do_run_mem;

    while (pim_running || mem_running) {
        thread_finished.acquire();

        if (pim_running && pim_finished.try_acquire()) {
            pim_running = false;
            std::cout << "<<< PIM FINISHED >>>" << std::endl;
        }

        if (mem_running && mem_finished.try_acquire()) {
            mem_running = false;
            std::cout << "<<< MEM FINISHED >>>" << std::endl;
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));

    if (do_run_pim) {
        free_pim(pim_qk);
        free_pim(pim_softmax);
        free_pim(pim_sv);
    }
}
