#include <cassert>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <mutex>
#include <thread>
#include "../../common.h"

#define MIN_ITERS 1
#define PIM_ROWS  256
#define LLM_NUM_MEM_KERNELS 3

class Semaphore {
    public:
        Semaphore (int count_ = 0) : count(count_) {}

        inline void notify()
        {
            mtx.lock();
            count++;
            mtx.unlock();
        }

        inline bool done()
        {
            bool retval = true;

            mtx.lock();
            if (count == 0) {
                retval = false;
            } else {
                count--;
            }
            mtx.unlock();

            return retval;
        }

private:
        std::mutex mtx;
        int count;
};

// Global state variables
int (*mem_app) (int, char**);
pim_state_t *pim_state;
hipStream_t pim_stream;
Semaphore pim_running;

// Rodinia benchmark declarations
extern "C" int main_btree(int argc, char** argv);
int main_backprop(int argc, char** argv);
int main_bfs(int argc, char** argv);
int main_euler3d(int argc, char** argv);  // CFD
int main_dwt2d(int argc, char** argv);
int main_gaussian(int argc, char** argv);
int main_heartwall(int argc, char** argv);
int main_hotspot(int argc, char** argv);
int main_hotspot3D(int argc, char** argv);
int main_huffman(int argc, char** argv);
int main_kmeans(int argc, char** argv);
int main_lavaMD(int argc, char** argv);
extern "C" int main_leukocyte(int argc, char** argv);
int main_lud(int argc, char** argv);
int main_mummergpu(int argc, char** argv);
int main_nn(int argc, char** argv, bool is_first);
int main_nw(int argc, char** argv);
int main_pathfinder(int argc, char** argv);
int main_srad_v1(int argc, char** argv);
int main_srad_v2(int argc, char** argv);
int main_streamcluster(int argc, char** argv);
void main_gemm(hipStream_t stream, size_t M, size_t K, size_t N);

// Helper functions
void setup_mem(char*);
void setup_pim(char *kernel);
void run_mem(char*, int, char**, Semaphore*, bool, bool);
void run_gemm(hipStream_t, Semaphore*, bool);
void run_pim(Semaphore*, bool);
void run_pim_llm(Semaphore*, pim_state_t*, pim_state_t*, pim_state_t*);
void exec_mem_and_pim(char*, char*, int, char**);
void exec_mem_only(char*, int, char**);
void exec_llm(bool, bool);

void print_usage(char *argv0)
{
    std::cout << "Usage: " << argv0 << "<pim> <mem>" << std::endl;
    std::cout << "       " << argv0 << " llm" << std::endl;
}

int main(int argc, char **argv)
{
    int error_code = 0;

    if (argc >= 3) {
        char *mem_app_name = argv[2];
        char *pim_app_name = argv[1];

        if (!strcmp(pim_app_name, "nop")) {
            exec_mem_only(mem_app_name, argc, argv);
        } else {
            exec_mem_and_pim(mem_app_name, pim_app_name, argc, argv);
        }
    }
    else if (argc == 2) {
        char *app_name = argv[1];

        if (!strcmp(app_name, "llm")) {
            exec_llm(true, true);
        } else if (!strcmp(app_name, "llm_mem_only")) {
            exec_llm(true, false);
        } else if (!strcmp(app_name, "llm_pim_only")) {
            exec_llm(false, true);
        } else {
            print_usage(argv[0]);
            error_code = -2;
        }
    }
    else {
        print_usage(argv[0]);
        error_code = -1;
    }

    return error_code;
}

void setup_mem(char *kernel)
{
    if (!strcmp(kernel, "b+tree")) {
        mem_app = main_btree;
    } else if (!strcmp(kernel, "backprop")) {
        mem_app = main_backprop;
    } else if (!strcmp(kernel, "bfs")) {
        mem_app = main_bfs;
    } else if (!strcmp(kernel, "cfd")) {
        mem_app = main_euler3d;
    } else if (!strcmp(kernel, "dwt2d")) {
        mem_app = main_dwt2d;
    } else if (!strcmp(kernel, "gaussian")) {
        mem_app = main_gaussian;
    } else if (!strcmp(kernel, "heartwall")) {
        mem_app = main_heartwall;
    } else if (!strcmp(kernel, "hotspot")) {
        mem_app = main_hotspot;
    } else if (!strcmp(kernel, "hotspot3D")) {
        mem_app = main_hotspot3D;
    } else if (!strcmp(kernel, "huffman")) {
        mem_app = main_huffman;
    } else if (!strcmp(kernel, "kmeans")) {
        mem_app = main_kmeans;
    } else if (!strcmp(kernel, "lavaMD")) {
        mem_app = main_lavaMD;
    } else if (!strcmp(kernel, "leukocyte")) {
        mem_app = main_leukocyte;
    } else if (!strcmp(kernel, "lud")) {
        mem_app = main_lud;
    } else if (!strcmp(kernel, "mummergpu")) {
        mem_app = main_mummergpu;
    } else if (!strcmp(kernel, "nn")) {
        // do nothing; nn needs special handling
    } else if (!strcmp(kernel, "nw")) {
        mem_app = main_nw;
    } else if (!strcmp(kernel, "pathfinder")) {
        mem_app = main_pathfinder;
    } else if (!strcmp(kernel, "srad_v1")) {
        mem_app = main_srad_v1;
    } else if (!strcmp(kernel, "srad_v2")) {
        mem_app = main_srad_v2;
    } else if (!strcmp(kernel, "streamcluster")) {
        mem_app = main_streamcluster;
    } else {
        std::cout << "Invalid MEM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void setup_pim(char *kernel)
{
    if (!strcmp(kernel, "stream_add")) {
        pim_state = init_pim(STREAM_ADD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_copy")) {
        pim_state = init_pim(STREAM_COPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_daxpy")) {
        pim_state = init_pim(STREAM_DAXPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_scale")) {
        pim_state = init_pim(STREAM_SCALE, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_triad")) {
        pim_state = init_pim(STREAM_TRIAD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_fwd")) {
        pim_state = init_pim(BN_FWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_bwd")) {
        pim_state = init_pim(BN_BWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "kmeans")) {
        pim_state = init_pim(KMEANS, 1048576, 1);
    } else if (!strcmp(kernel, "histogram")) {
        pim_state = init_pim(HISTOGRAM, 1048576, 1);
    } else if (!strcmp(kernel, "fully_connected")) {
        pim_state = init_pim(FULLY_CONNECTED, 1048576, 256);
    } else if (!strcmp(kernel, "grim")) {
        pim_state = init_pim(GRIM, 1048576, 32);
    } else {
        std::cout << "Invalid PIM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void run_mem(char *kernel, int argc, char **argv, Semaphore *semaphore,
        bool is_first, bool wait_for_pim)
{
    // create a copy of argv because some benchmarks destroy the arguments
    char **argv_copy = new char*[(argc - 2) + 1];
    for (int i = 2; i < argc; i++) {
        argv_copy[i - 2] = new char[strlen(argv[i]) + 1];
        strcpy(argv_copy[i - 2], argv[i]);
    }
    argv_copy[argc - 2] = NULL;

    if (wait_for_pim) { while (!pim_running.done()); }

    if (!strcmp(kernel, "nn")) {
        main_nn(argc - 2, argv_copy, is_first);
    } else {
        mem_app(argc - 2, argv_copy);
    }

    hipStreamSynchronize(0);
    semaphore->notify();

    // we don't need to destroy argv_copy because:
    // 1) if the benchmark modified the pointer, freeing it can cause a
    //    segfault
    // 2) the thread is being destroyed right now, so the memory will be freed
    //    anyway
}

void run_gemm(hipStream_t stream, Semaphore *semaphore, bool wait_for_pim)
{
    if (wait_for_pim) {
        while (!pim_running.done());
    }

    for (int i = 0; i < LLM_NUM_MEM_KERNELS; i++) {
        main_gemm(stream, 128, 4096, 4096);
        hipStreamSynchronize(stream);
    }

    semaphore->notify();
}

void run_pim(Semaphore *semaphore, bool is_first)
{
    launch_pim(pim_state, pim_stream);
    if (is_first) {
        // wait for PIM to start running
        hipStreamGetPriority(pim_stream, NULL);
        pim_running.notify();
    }
    hipStreamSynchronize(pim_stream);
    semaphore->notify();
}

void run_pim_llm(Semaphore *semaphore, pim_state_t *pim_qk,
        pim_state_t *pim_softmax, pim_state_t *pim_sv)
{
    launch_pim(pim_qk,      pim_stream);
    launch_pim(pim_softmax, pim_stream);
    launch_pim(pim_sv,      pim_stream);

    // wait for PIM to start running
    hipStreamGetPriority(pim_stream, NULL);
    pim_running.notify();

    hipStreamSynchronize(pim_stream);
    semaphore->notify();
}

void exec_mem_and_pim(char *mem_app_name, char *pim_app_name, int argc,
        char **argv)
{
    setup_mem(mem_app_name);
    setup_pim(pim_app_name);

    hipStreamCreateWithPriority(&pim_stream, 0, -1);

    unsigned mem_iters = 0, pim_iters = 0;
    bool mem_running = false, pim_running = false;
    Semaphore mem_semaphore, pim_semaphore;

    while ((mem_iters < MIN_ITERS) || (pim_iters < MIN_ITERS)) {
        if (!mem_running && !pim_running) {
            std::thread (run_pim, &pim_semaphore, true).detach();
            std::thread (run_mem, mem_app_name, argc, argv, &mem_semaphore,
                    true, true).detach();

            mem_running = true;
            pim_running = true;
        }

        else if (!mem_running) {
            std::thread (run_mem, mem_app_name, argc, argv, &mem_semaphore,
                    false, false).detach();
            mem_running = true;
        }

        else if (!pim_running) {
            std::thread (run_pim, &pim_semaphore, false).detach();
            pim_running = true;
        }

        while (true) {
            if (mem_semaphore.done()) {
                mem_iters++;
                mem_running = false;
                std::cout << "<<< MEM FINISHED >>>" << std::endl;
                hipGetErrorName(hipSuccess);
                break;
            }

            if (pim_semaphore.done()) {
                pim_iters++;
                pim_running = false;
                std::cout << "<<< PIM FINISHED >>>" << std::endl;
                hipGetErrorName(hipSuccess);
                break;
            }
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));

    free_pim(pim_state);
}

void exec_mem_only(char *mem_app_name, int argc, char **argv)
{
    setup_mem(mem_app_name);

    unsigned mem_iters = 0;
    Semaphore mem_semaphore;

    while (mem_iters < MIN_ITERS) {
        std::thread (run_mem, mem_app_name, argc, argv, &mem_semaphore,
                mem_iters == 0, false).detach();

        while (!mem_semaphore.done());
        mem_iters++;
        std::cout << "<<< MEM FINISHED >>>" << std::endl;
        hipGetErrorName(hipSuccess);
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
}

void exec_llm(bool run_mem, bool run_pim)
{
    assert(run_mem || run_pim);

    pim_state_t *pim_qk, *pim_softmax, *pim_sv;
    hipStream_t mem_stream;

    if (run_pim) {
        pim_qk = init_pim(FULLY_CONNECTED, 1048576, 1024);
        pim_softmax = init_pim(SOFTMAX, 1048576, 128);
        pim_sv = init_pim(FULLY_CONNECTED_128_ELEM, 1048576, 1024);

        // Higher priority stream for PIM
        hipStreamCreateWithPriority(&pim_stream, 0, -1);
    }

    if (run_mem) {
        // Regular priority stream for MEM
        hipStreamCreate(&mem_stream);
    }

    Semaphore pim_semaphore, mem_semaphore;

    if (run_pim) {
        std::thread (run_pim_llm, &pim_semaphore, pim_qk, pim_softmax,
                pim_sv).detach();
    }

    if (run_mem) {
        std::thread (run_gemm, mem_stream, &mem_semaphore, run_pim).detach();
    }

    bool pim_running = run_pim;
    bool mem_running = run_mem;

    while (pim_running || mem_running) {
        if (pim_running && pim_semaphore.done()) {
            pim_running = false;
            std::cout << "<<< PIM FINISHED >>>" << std::endl;
        }

        if (mem_running && mem_semaphore.done()) {
            mem_running = false;
            std::cout << "<<< MEM FINISHED >>>" << std::endl;
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));

    if (run_pim) {
        free_pim(pim_qk);
        free_pim(pim_softmax);
        free_pim(pim_sv);
    }
}
