#include <chrono>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <mutex>
#include <thread>
#include "../../common.h"

#define MIN_ITERS 1
#define PIM_ROWS  256

class Semaphore {
    public:
        Semaphore (int count_ = 0) : count(count_) {}

        inline void notify()
        {
            mtx.lock();
            count++;
            mtx.unlock();
        }

        inline bool done()
        {
            bool retval = true;

            mtx.lock();
            if (count == 0) {
                retval = false;
            } else {
                count--;
            }
            mtx.unlock();

            return retval;
        }

private:
        std::mutex mtx;
        int count;
};

// Global state variables
int (*mem_app) (int, char**);
pim_state_t *pim_state;
hipStream_t pim_stream;
Semaphore pim_running;

// Rodinia benchmark declarations
extern "C" int main_btree(int argc, char** argv);
int main_backprop(int argc, char** argv);
int main_bfs(int argc, char** argv);
int main_euler3d(int argc, char** argv);  // CFD
int main_dwt2d(int argc, char** argv);
int main_gaussian(int argc, char** argv);
int main_heartwall(int argc, char** argv);
int main_hotspot(int argc, char** argv);
int main_hotspot3D(int argc, char** argv);
int main_huffman(int argc, char** argv);
int main_kmeans(int argc, char** argv);
int main_lavaMD(int argc, char** argv);
extern "C" int main_leukocyte(int argc, char** argv);
int main_lud(int argc, char** argv);
int main_mummergpu(int argc, char** argv);
int main_nn(int argc, char** argv, bool is_first);
int main_nw(int argc, char** argv);
int main_pathfinder(int argc, char** argv);
int main_srad_v1(int argc, char** argv);
int main_srad_v2(int argc, char** argv);
int main_streamcluster(int argc, char** argv);

// Helper functions
void setup_mem(char*);
void setup_pim(char *kernel);
void run_mem(char*, int, char**, Semaphore*, bool, bool);
void run_pim(Semaphore*, bool);
void exec_mem_and_pim(char*, char*, int, char**);
void exec_mem_only(char*, int, char**);

int main(int argc, char **argv)
{
    if (argc < 3) {
        std::cout << "Usage: " << argv[0] << "<pim> <mem>" << std::endl;
        return -1;
    }

    char *mem_app_name = argv[2];
    char *pim_app_name = argv[1];

    if (!strcmp(pim_app_name, "nop")) {
        exec_mem_only(mem_app_name, argc, argv);
    } else {
        exec_mem_and_pim(mem_app_name, pim_app_name, argc, argv);
    }

    return 0;
}

void setup_mem(char *kernel)
{
    if (!strcmp(kernel, "b+tree")) {
        mem_app = main_btree;
    } else if (!strcmp(kernel, "backprop")) {
        mem_app = main_backprop;
    } else if (!strcmp(kernel, "bfs")) {
        mem_app = main_bfs;
    } else if (!strcmp(kernel, "cfd")) {
        mem_app = main_euler3d;
    } else if (!strcmp(kernel, "dwt2d")) {
        mem_app = main_dwt2d;
    } else if (!strcmp(kernel, "gaussian")) {
        mem_app = main_gaussian;
    } else if (!strcmp(kernel, "heartwall")) {
        mem_app = main_heartwall;
    } else if (!strcmp(kernel, "hotspot")) {
        mem_app = main_hotspot;
    } else if (!strcmp(kernel, "hotspot3D")) {
        mem_app = main_hotspot3D;
    } else if (!strcmp(kernel, "huffman")) {
        mem_app = main_huffman;
    } else if (!strcmp(kernel, "kmeans")) {
        mem_app = main_kmeans;
    } else if (!strcmp(kernel, "lavaMD")) {
        mem_app = main_lavaMD;
    } else if (!strcmp(kernel, "leukocyte")) {
        mem_app = main_leukocyte;
    } else if (!strcmp(kernel, "lud")) {
        mem_app = main_lud;
    } else if (!strcmp(kernel, "mummergpu")) {
        mem_app = main_mummergpu;
    } else if (!strcmp(kernel, "nn")) {
        // do nothing; nn needs special handling
    } else if (!strcmp(kernel, "nw")) {
        mem_app = main_nw;
    } else if (!strcmp(kernel, "pathfinder")) {
        mem_app = main_pathfinder;
    } else if (!strcmp(kernel, "srad_v1")) {
        mem_app = main_srad_v1;
    } else if (!strcmp(kernel, "srad_v2")) {
        mem_app = main_srad_v2;
    } else if (!strcmp(kernel, "streamcluster")) {
        mem_app = main_streamcluster;
    } else {
        std::cout << "Invalid MEM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void setup_pim(char *kernel)
{
    if (!strcmp(kernel, "stream_add")) {
        pim_state = init_pim(STREAM_ADD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_copy")) {
        pim_state = init_pim(STREAM_COPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_daxpy")) {
        pim_state = init_pim(STREAM_DAXPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_scale")) {
        pim_state = init_pim(STREAM_SCALE, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_triad")) {
        pim_state = init_pim(STREAM_TRIAD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_fwd")) {
        pim_state = init_pim(BN_FWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_bwd")) {
        pim_state = init_pim(BN_BWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "kmeans")) {
        pim_state = init_pim(KMEANS, 1048576, 1);
    } else if (!strcmp(kernel, "histogram")) {
        pim_state = init_pim(HISTOGRAM, 1048576, 1);
    } else if (!strcmp(kernel, "fully_connected")) {
        pim_state = init_pim(FULLY_CONNECTED, 1048576, 1);
    } else if (!strcmp(kernel, "grim")) {
        pim_state = init_pim(GRIM, 1048576, 32);
    } else {
        std::cout << "Invalid PIM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void run_mem(char *kernel, int argc, char **argv, Semaphore *semaphore,
        bool is_first, bool wait_for_pim)
{
    // create a copy of argv because some benchmarks destroy the arguments
    char **argv_copy = new char*[(argc - 2) + 1];
    for (int i = 2; i < argc; i++) {
        argv_copy[i - 2] = new char[strlen(argv[i])];
        strcpy(argv_copy[i - 2], argv[i]);
    }
    argv_copy[argc - 2] = NULL;

    if (wait_for_pim) { while(!pim_running.done()); }

    if (!strcmp(kernel, "nn")) {
        main_nn(argc - 2, argv_copy, is_first);
    } else {
        mem_app(argc - 2, argv_copy);
    }

    hipStreamSynchronize(0);
    semaphore->notify();

    // we don't need to destroy argv_copy because:
    // 1) if the benchmark modified the pointer, freeing it can cause a
    //    segfault
    // 2) the thread is being destroyed right now, so the memory will be freed
    //    anyway
}

void run_pim(Semaphore *semaphore, bool is_first)
{
    launch_pim(pim_state, pim_stream);
    if (is_first) {
        // wait for PIM to start running
        hipStreamGetPriority(pim_stream, NULL);
        pim_running.notify();
    }
    hipStreamSynchronize(pim_stream);
    semaphore->notify();
}

void exec_mem_and_pim(char *mem_app_name, char *pim_app_name, int argc,
        char **argv)
{
    setup_mem(mem_app_name);
    setup_pim(pim_app_name);

    hipStreamCreate(&pim_stream);

    unsigned mem_iters = 0, pim_iters = 0;
    bool mem_running = false, pim_running = false;
    Semaphore mem_semaphore, pim_semaphore;

    while ((mem_iters < MIN_ITERS) || (pim_iters < MIN_ITERS)) {
        if (!mem_running && !pim_running) {
            std::thread (run_pim, &pim_semaphore, true).detach();
            std::thread (run_mem, mem_app_name, argc, argv, &mem_semaphore,
                    true, true).detach();

            mem_running = true;
            pim_running = true;
        }

        else if (!mem_running) {
            std::thread (run_mem, mem_app_name, argc, argv, &mem_semaphore,
                    false, false).detach();
            mem_running = true;
        }

        else if (!pim_running) {
            std::thread (run_pim, &pim_semaphore, false).detach();
            pim_running = true;
        }

        while (true) {
            if (mem_semaphore.done()) {
                mem_iters++;
                mem_running = false;
                std::cout << "<<< MEM FINISHED >>>" << std::endl;
                hipGetErrorName(hipSuccess);
                break;
            }

            if (pim_semaphore.done()) {
                pim_iters++;
                pim_running = false;
                std::cout << "<<< PIM FINISHED >>>" << std::endl;
                hipGetErrorName(hipSuccess);
                break;
            }
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));

    free_pim(pim_state);
}

void exec_mem_only(char *mem_app_name, int argc, char **argv)
{
    setup_mem(mem_app_name);

    unsigned mem_iters = 0;
    Semaphore mem_semaphore;

    while (mem_iters < MIN_ITERS) {
        std::thread (run_mem, mem_app_name, argc, argv, &mem_semaphore,
                mem_iters == 0, false).detach();

        while (!mem_semaphore.done());
        mem_iters++;
        std::cout << "<<< MEM FINISHED >>>" << std::endl;
        hipGetErrorName(hipSuccess);
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
}
