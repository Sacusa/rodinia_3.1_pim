#include <cassert>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <thread>
#include "../../common.h"

/**
 * std::semaphore support was added in C++20 and requires CUDA 12+. We provide
 * a comparable implementation for older versions of CUDA, such as used the one
 * we use. This requires C++11 support.
 */

#if __cplusplus >= 202002L

#include <semaphore>
using Semaphore std::counting_semaphore;

#else

#include <condition_variable>
#include <mutex>

class Semaphore {
    public:
        Semaphore (unsigned count_ = 0) : count(count_) {}

        inline void acquire() {
            std::unique_lock<std::mutex> lock(mtx);
            if (count == 0) {
                cv.wait(lock, [&] { return count > 0; });
            }
            --count;
        }

        inline bool try_acquire() {
            std::unique_lock<std::mutex> lock(mtx);
            if (count == 0) {
                return false;
            } else {
                --count;
                return true;
            }
        }

        inline void release() {
            std::unique_lock<std::mutex> lock(mtx);
            ++count;
            lock.unlock();
            cv.notify_all();
        }

private:
        std::mutex mtx;
        std::condition_variable cv;
        unsigned count;
};

#endif

#define MIN_ITERS 1
#define PIM_ROWS  256
#define LLM_NUM_MEM_KERNELS 3

// Global state variables
int (*mem_app) (int, char**);
pim_state_t *pim_state;
hipStream_t pim_stream;
Semaphore thread_finished{0};

// Rodinia benchmark declarations
extern "C" int main_btree(int argc, char** argv);
int main_backprop(int argc, char** argv);
int main_bfs(int argc, char** argv);
int main_euler3d(int argc, char** argv);  // CFD
int main_dwt2d(int argc, char** argv);
int main_gaussian(int argc, char** argv);
int main_heartwall(int argc, char** argv);
int main_hotspot(int argc, char** argv);
int main_hotspot3D(int argc, char** argv);
int main_huffman(int argc, char** argv);
int main_kmeans(int argc, char** argv);
int main_lavaMD(int argc, char** argv);
extern "C" int main_leukocyte(int argc, char** argv);
int main_lud(int argc, char** argv);
int main_mummergpu(int argc, char** argv);
int main_nn(int argc, char** argv, bool is_first);
int main_nw(int argc, char** argv);
int main_pathfinder(int argc, char** argv);
int main_srad_v1(int argc, char** argv);
int main_srad_v2(int argc, char** argv);
int main_streamcluster(int argc, char** argv);
void main_gemm(hipStream_t stream, size_t M, size_t K, size_t N);

// Helper functions
void setup_mem(char*);
void setup_pim(char *kernel);
void run_mem(char*, int, char**, bool, bool, Semaphore*, Semaphore*);
void run_gemm(hipStream_t, bool, Semaphore*, Semaphore*);
void run_pim(bool, Semaphore*, Semaphore*);
void run_pim_llm(pim_state_t*, pim_state_t*, pim_state_t*, Semaphore*,
        Semaphore*);
void exec_mem_and_pim(char*, char*, int, char**);
void exec_mem_only(char*, int, char**);
void exec_llm(bool, bool);

void print_usage(char *argv0)
{
    std::cout << "Usage: " << argv0 << "<pim> <mem>" << std::endl;
    std::cout << "       " << argv0 << " llm" << std::endl;
}

int main(int argc, char **argv)
{
    int error_code = 0;

    if (argc >= 3) {
        char *mem_app_name = argv[2];
        char *pim_app_name = argv[1];

        if (!strcmp(pim_app_name, "nop")) {
            exec_mem_only(mem_app_name, argc, argv);
        } else {
            exec_mem_and_pim(mem_app_name, pim_app_name, argc, argv);
        }
    }
    else if (argc == 2) {
        char *app_name = argv[1];

        if (!strcmp(app_name, "llm")) {
            exec_llm(true, true);
        } else if (!strcmp(app_name, "llm_mem_only")) {
            exec_llm(true, false);
        } else if (!strcmp(app_name, "llm_pim_only")) {
            exec_llm(false, true);
        } else {
            print_usage(argv[0]);
            error_code = -2;
        }
    }
    else {
        print_usage(argv[0]);
        error_code = -1;
    }

    return error_code;
}

void setup_mem(char *kernel)
{
    if (!strcmp(kernel, "b+tree")) {
        mem_app = main_btree;
    } else if (!strcmp(kernel, "backprop")) {
        mem_app = main_backprop;
    } else if (!strcmp(kernel, "bfs")) {
        mem_app = main_bfs;
    } else if (!strcmp(kernel, "cfd")) {
        mem_app = main_euler3d;
    } else if (!strcmp(kernel, "dwt2d")) {
        mem_app = main_dwt2d;
    } else if (!strcmp(kernel, "gaussian")) {
        mem_app = main_gaussian;
    } else if (!strcmp(kernel, "heartwall")) {
        mem_app = main_heartwall;
    } else if (!strcmp(kernel, "hotspot")) {
        mem_app = main_hotspot;
    } else if (!strcmp(kernel, "hotspot3D")) {
        mem_app = main_hotspot3D;
    } else if (!strcmp(kernel, "huffman")) {
        mem_app = main_huffman;
    } else if (!strcmp(kernel, "kmeans")) {
        mem_app = main_kmeans;
    } else if (!strcmp(kernel, "lavaMD")) {
        mem_app = main_lavaMD;
    } else if (!strcmp(kernel, "leukocyte")) {
        mem_app = main_leukocyte;
    } else if (!strcmp(kernel, "lud")) {
        mem_app = main_lud;
    } else if (!strcmp(kernel, "mummergpu")) {
        mem_app = main_mummergpu;
    } else if (!strcmp(kernel, "nn")) {
        // do nothing; nn needs special handling
    } else if (!strcmp(kernel, "nw")) {
        mem_app = main_nw;
    } else if (!strcmp(kernel, "pathfinder")) {
        mem_app = main_pathfinder;
    } else if (!strcmp(kernel, "srad_v1")) {
        mem_app = main_srad_v1;
    } else if (!strcmp(kernel, "srad_v2")) {
        mem_app = main_srad_v2;
    } else if (!strcmp(kernel, "streamcluster")) {
        mem_app = main_streamcluster;
    } else {
        std::cout << "Invalid MEM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void setup_pim(char *kernel)
{
    if (!strcmp(kernel, "stream_add")) {
        pim_state = init_pim(STREAM_ADD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_copy")) {
        pim_state = init_pim(STREAM_COPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_daxpy")) {
        pim_state = init_pim(STREAM_DAXPY, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_scale")) {
        pim_state = init_pim(STREAM_SCALE, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "stream_triad")) {
        pim_state = init_pim(STREAM_TRIAD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_fwd")) {
        pim_state = init_pim(BN_FWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "bn_bwd")) {
        pim_state = init_pim(BN_BWD, 1048576, PIM_ROWS);
    } else if (!strcmp(kernel, "kmeans")) {
        pim_state = init_pim(KMEANS, 1048576, 1);
    } else if (!strcmp(kernel, "histogram")) {
        pim_state = init_pim(HISTOGRAM, 1048576, 1);
    } else if (!strcmp(kernel, "fully_connected")) {
        pim_state = init_pim(FULLY_CONNECTED, 1048576, 256);
    } else if (!strcmp(kernel, "grim")) {
        pim_state = init_pim(GRIM, 1048576, 32);
    } else {
        std::cout << "Invalid PIM application: " << kernel << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void run_mem(char *kernel, int argc, char **argv, bool wait_for_pim,
        bool is_first, Semaphore *pim_running, Semaphore *signal_mem_finished)
{
    // create a copy of argv because some benchmarks destroy the arguments
    char **argv_copy = new char*[(argc - 2) + 1];
    for (int i = 2; i < argc; i++) {
        argv_copy[i - 2] = new char[strlen(argv[i]) + 1];
        strcpy(argv_copy[i - 2], argv[i]);
    }
    argv_copy[argc - 2] = NULL;

    if (wait_for_pim) { pim_running->acquire(); }

    if (!strcmp(kernel, "nn")) {
        main_nn(argc - 2, argv_copy, is_first);
    } else {
        mem_app(argc - 2, argv_copy);
    }

    hipStreamSynchronize(0);
    signal_mem_finished->release();
    thread_finished.release();

    // we don't need to destroy argv_copy because:
    // 1) if the benchmark modified the pointer, freeing it can cause a
    //    segfault
    // 2) the thread is being destroyed right now, so the memory will be freed
    //    anyway
}

void run_gemm(hipStream_t stream, bool wait_for_pim,
        Semaphore *pim_running, Semaphore *signal_mem_finished)
{
    if (wait_for_pim) { pim_running->acquire(); }

    for (int i = 0; i < LLM_NUM_MEM_KERNELS; i++) {
        main_gemm(stream, 128, 4096, 4096);
        hipStreamSynchronize(stream);
    }

    signal_mem_finished->release();
    thread_finished.release();
}

void run_pim(bool do_signal_launch, Semaphore *pim_running,
        Semaphore *signal_pim_finished)
{
    launch_pim(pim_state, pim_stream);

    if (do_signal_launch) { pim_running->release(); }

    hipStreamSynchronize(pim_stream);
    signal_pim_finished->release();
    thread_finished.release();
}

void run_pim_llm(pim_state_t *pim_qk, pim_state_t *pim_softmax,
        pim_state_t *pim_sv, Semaphore *pim_running,
        Semaphore *signal_pim_finished)
{
    launch_pim(pim_qk,      pim_stream);
    launch_pim(pim_softmax, pim_stream);
    launch_pim(pim_sv,      pim_stream);

    pim_running->release();

    hipStreamSynchronize(pim_stream);
    signal_pim_finished->release();
    thread_finished.release();
}

void exec_mem_and_pim(char *mem_app_name, char *pim_app_name, int argc,
        char **argv)
{
    setup_mem(mem_app_name);
    setup_pim(pim_app_name);

    hipStreamCreateWithPriority(&pim_stream, 0, -1);

    unsigned mem_iters = 0, pim_iters = 0;
    bool mem_running = false, pim_running = false;
    Semaphore pim_launched{0}, mem_finished{0}, pim_finished{0};

    while ((mem_iters < MIN_ITERS) || (pim_iters < MIN_ITERS)) {
        if (!mem_running && !pim_running) {
            std::thread (run_pim, true, &pim_launched,
                    &pim_finished).detach();
            std::thread (run_mem, mem_app_name, argc, argv, true, true,
                    &pim_launched, &mem_finished).detach();

            mem_running = true;
            pim_running = true;
        }

        else if (!mem_running) {
            std::thread (run_mem, mem_app_name, argc, argv, false, false,
                    &pim_launched, &mem_finished).detach();
            mem_running = true;
        }

        else if (!pim_running) {
            std::thread (run_pim, false, &pim_launched,
                    &pim_finished).detach();
            pim_running = true;
        }

        thread_finished.acquire();

        if (mem_finished.try_acquire()) {
            mem_iters++;
            mem_running = false;
            std::cout << "<<< MEM FINISHED >>>" << std::endl;
            hipGetErrorName(hipSuccess);
        }

        else if (pim_finished.try_acquire()) {
            pim_iters++;
            pim_running = false;
            std::cout << "<<< PIM FINISHED >>>" << std::endl;
            hipGetErrorName(hipSuccess);
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));

    free_pim(pim_state);
}

void exec_mem_only(char *mem_app_name, int argc, char **argv)
{
    setup_mem(mem_app_name);

    unsigned mem_iters = 0;
    Semaphore mem_finished{0};

    while (mem_iters < MIN_ITERS) {
        std::thread (run_mem, mem_app_name, argc, argv, false, mem_iters == 0,
                nullptr, &mem_finished).detach();

        thread_finished.acquire();

        mem_finished.acquire();
        mem_iters++;
        std::cout << "<<< MEM FINISHED >>>" << std::endl;
        hipGetErrorName(hipSuccess);
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
}

void exec_llm(bool do_run_mem, bool do_run_pim)
{
    assert(do_run_mem || do_run_pim);

    pim_state_t *pim_qk, *pim_softmax, *pim_sv;
    hipStream_t mem_stream;

    if (do_run_pim) {
        pim_qk = init_pim(FULLY_CONNECTED, 1048576, 1024);
        pim_softmax = init_pim(SOFTMAX, 1048576, 128);
        pim_sv = init_pim(FULLY_CONNECTED_128_ELEM, 1048576, 1024);

        // Higher priority stream for PIM
        hipStreamCreateWithPriority(&pim_stream, 0, -1);
    }

    if (do_run_mem) {
        // Regular priority stream for MEM
        hipStreamCreate(&mem_stream);
    }

    Semaphore pim_launched{0}, mem_finished{0}, pim_finished{0};

    if (do_run_pim) {
        std::thread (run_pim_llm, pim_qk, pim_softmax, pim_sv, &pim_launched,
                &pim_finished).detach();
    }

    if (do_run_mem) {
        std::thread (run_gemm, mem_stream, do_run_pim, &pim_launched,
                &mem_finished).detach();
    }

    bool pim_running = do_run_pim;
    bool mem_running = do_run_mem;

    while (pim_running || mem_running) {
        thread_finished.acquire();

        if (pim_running && pim_finished.try_acquire()) {
            pim_running = false;
            std::cout << "<<< PIM FINISHED >>>" << std::endl;
        }

        if (mem_running && mem_finished.try_acquire()) {
            mem_running = false;
            std::cout << "<<< MEM FINISHED >>>" << std::endl;
        }
    }

    // Kill all running kernels
    hipDeviceReset();

    // Sleep for a second so that GPGPU-Sim can clean up
    std::this_thread::sleep_for(std::chrono::milliseconds(1000));

    if (do_run_pim) {
        free_pim(pim_qk);
        free_pim(pim_softmax);
        free_pim(pim_sv);
    }
}
